#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void testKernel() {
    printf("Hello from GPU!\n");
}

int main() {
    printf("CUDA test program\n");
    
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    
    if (deviceCount == 0) {
        printf("No CUDA devices found!\n");
        return 1;
    }
    
    printf("Found %d CUDA device(s)\n", deviceCount);
    
    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device %d: %s\n", i, prop.name);
    }
    
    testKernel<<<1, 1>>>();
    hipDeviceSynchronize();
    
    return 0;
}